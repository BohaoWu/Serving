// Copyright (c) 2020 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

/*
* Copyright 2017-2018 NVIDIA Corporation.  All rights reserved.
*
* Please refer to the NVIDIA end user license agreement (EULA) associated
* with this source code for terms and conditions that govern your use of
* this software. Any use, reproduction, disclosure, or distribution of
* this software and related documentation outside the terms of the EULA
* is strictly prohibited.
*
*/

#include <hip/hip_runtime.h>
#include <stdint.h>
#include <stdio.h>

static __global__ void ConvertUInt8ToUInt16Kernel(uint8_t *dpUInt8, uint16_t *dpUInt16, int nSrcPitch, int nDestPitch, int nWidth, int nHeight)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x,
        y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= nWidth || y >= nHeight)
    {
        return;
    }
    int destStrideInPixels = nDestPitch / (sizeof(uint16_t));
    *(uchar2 *)&dpUInt16[y * destStrideInPixels + x] = uchar2{ 0, dpUInt8[y * nSrcPitch + x] };
}

static __global__ void ConvertUInt16ToUInt8Kernel(uint16_t *dpUInt16, uint8_t *dpUInt8, int nSrcPitch, int nDestPitch, int nWidth, int nHeight)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x,
        y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= nWidth || y >= nHeight)
    {
        return;
    }
    int srcStrideInPixels = nSrcPitch / (sizeof(uint16_t));
    dpUInt8[y * nDestPitch + x] = ((uchar2 *)&dpUInt16[y * srcStrideInPixels + x])->y;
}

void ConvertUInt8ToUInt16(uint8_t *dpUInt8, uint16_t *dpUInt16, int nSrcPitch, int nDestPitch, int nWidth, int nHeight)
{
    dim3 blockSize(16, 16, 1);
    dim3 gridSize(((uint32_t)nWidth + blockSize.x - 1) / blockSize.x, ((uint32_t)nHeight + blockSize.y - 1) / blockSize.y, 1);
    ConvertUInt8ToUInt16Kernel <<< gridSize, blockSize >>>(dpUInt8, dpUInt16, nSrcPitch, nDestPitch, nWidth, nHeight);
}

void ConvertUInt16ToUInt8(uint16_t *dpUInt16, uint8_t *dpUInt8, int nSrcPitch, int nDestPitch, int nWidth, int nHeight)
{
    dim3 blockSize(16, 16, 1);
    dim3 gridSize(((uint32_t)nWidth + blockSize.x - 1) / blockSize.x, ((uint32_t)nHeight + blockSize.y - 1) / blockSize.y, 1);
    ConvertUInt16ToUInt8Kernel <<<gridSize, blockSize >>>(dpUInt16, dpUInt8, nSrcPitch, nDestPitch, nWidth, nHeight);
}
